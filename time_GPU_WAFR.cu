#include "hip/hip_runtime.h"
/***
nvcc -std=c++11 -o WAFR_timing.exe time_GPU_WAFR.cu -gencode arch=compute_75,code=sm_75 -O3
***/

#include "utils/experiment_helpers.h" // include constants and other experiment consistency helpers
#include "helpers_WAFR/dynamics_WAFR.cuh" // for GPU dynamicsGradient

#if TEST_FOR_EQUIVALENCE
	dim3 dimms(1,1);
#else
	dim3 dimms(16,16);
#endif

template<typename T, int TEST_ITERS, int NUM_TIME_STEPS_TEST, bool VEL_DAMPING = false>
void test(){
	// allocate and load on CPU
	T *h_qdd = (T *)malloc(NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T));
	T *h_dqdd = (T *)malloc(3*NUM_POS*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T));
	T *h_dqdd2 = (T *)malloc(3*NUM_POS*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T));
	T *h_Minv = (T *)malloc(NUM_POS*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T));
	T *h_x = (T *)malloc(STATE_SIZE*NUM_TIME_STEPS_TEST*sizeof(T));
	T *h_u = (T *)malloc(CONTROL_SIZE*NUM_TIME_STEPS_TEST*sizeof(T));
	T *h_I = (T *)malloc(36*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)); initI(h_I);
	T *h_T = (T *)malloc(36*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)); initT(h_T);
	T *h_mem_vol = (T *)malloc((4*NUM_POS + NUM_POS*NUM_POS)*NUM_TIME_STEPS_TEST*sizeof(T)); // x,u,qdd,Minv
	T *h_mem_const = (T *)malloc(72*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)); // I,Tbody
	#pragma unroll
	for (int k = 0; k < NUM_TIME_STEPS_TEST; k++){
		T *h_qddk = &h_qdd[k*NUM_POS];
		T *h_xk = &h_x[k*2*NUM_POS];
		T *h_uk = &h_u[k*NUM_POS];
		T *h_Minvk = &h_Minv[k*NUM_POS*NUM_POS];
		T *h_mem_volk = &h_mem_vol[k*(4*NUM_POS + NUM_POS*NUM_POS)];
		#if TEST_FOR_EQUIVALENCE
			for(int j = 0; j < NUM_POS; j++){h_xk[j] = 0.1; h_xk[j+NUM_POS] = 0.1; h_uk[j] = 0.1;}
		#else
			for(int i = 0; i < NUM_POS; i++){h_xk[i] = getRand<T>(); h_xk[i+NUM_POS] = getRand<T>(); h_uk[i] = getRand<T>();}
		#endif
		dynamicsMinv<T,VEL_DAMPING>(h_qddk,h_xk,h_uk,h_I,h_T,h_Minvk);
		for(int i = 0; i < 4*NUM_POS + NUM_POS*NUM_POS; i++){
			if (i < 2*NUM_POS){h_mem_volk[i] = h_xk[i];}
			else if (i < 3*NUM_POS){h_mem_volk[i] = h_uk[i-2*NUM_POS];}
			else if (i < 4*NUM_POS){h_mem_volk[i] = h_qddk[i-3*NUM_POS];}
			else {h_mem_vol[i] = h_Minvk[i-4*NUM_POS];}
		}
	}
	memcpy(h_mem_const,h_I,36*NUM_POS*sizeof(T));
	memcpy(&h_mem_const[36*NUM_POS],h_T,36*NUM_POS*sizeof(T));

	// allocate and copy to GPU
	T *d_dqdd; gpuErrchk(hipMalloc((void**)&d_dqdd, 3*NUM_POS*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)));
	T *d_dqdd2; gpuErrchk(hipMalloc((void**)&d_dqdd2, 3*NUM_POS*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)));
	T *d_Minv; gpuErrchk(hipMalloc((void**)&d_Minv, NUM_POS*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)));
	T *d_x; gpuErrchk(hipMalloc((void**)&d_x, 2*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)));
	T *d_u; gpuErrchk(hipMalloc((void**)&d_u, NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)));
	T *d_qdd; gpuErrchk(hipMalloc((void**)&d_qdd, NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)));
	T *d_I; gpuErrchk(hipMalloc((void**)&d_I, 36*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)));
	T *d_T; gpuErrchk(hipMalloc((void**)&d_T, 36*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T)));
	T *d_mem_vol; gpuErrchk(hipMalloc((void**)&d_mem_vol, (4*NUM_POS + NUM_POS*NUM_POS)*NUM_TIME_STEPS_TEST*sizeof(T)));
	T *d_mem_const; gpuErrchk(hipMalloc((void**)&d_mem_const, 72*NUM_POS*sizeof(T)));
	hipStream_t *streams = (hipStream_t *)malloc(NUM_STREAMS*sizeof(hipStream_t));
	int priority, minPriority, maxPriority;
	gpuErrchk(hipDeviceGetStreamPriorityRange(&minPriority, &maxPriority));
	for(int i=0; i<NUM_STREAMS; i++){priority = std::min(minPriority+i,maxPriority);
		gpuErrchk(hipStreamCreateWithPriority(&(streams[i]),hipStreamNonBlocking,priority));
	}
	gpuErrchk(hipMemcpy(d_I,h_I,36*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_T,h_T,36*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_mem_const,h_mem_const,72*NUM_POS*sizeof(T),hipMemcpyHostToDevice));
	gpuErrchk(hipDeviceSynchronize());

	// time kernel with uncompressed and compressed memory copy
	struct timespec start, end;
	if(NUM_TIME_STEPS_TEST == 1){
		#if TEST_FOR_EQUIVALENCE
			#define SINGLE_TEST_ITERS (TEST_ITERS*1)
		#else
			#define SINGLE_TEST_ITERS (TEST_ITERS*10)
		#endif
		gpuErrchk(hipMemcpyAsync(d_Minv,h_Minv,NUM_POS*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T),hipMemcpyHostToDevice, streams[0]));
		gpuErrchk(hipMemcpyAsync(d_x,h_x,2*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T),hipMemcpyHostToDevice, streams[1]));
		gpuErrchk(hipMemcpyAsync(d_u,h_u,NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T),hipMemcpyHostToDevice, streams[2]));
		gpuErrchk(hipMemcpyAsync(d_qdd,h_qdd,NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T),hipMemcpyHostToDevice, streams[3]));
		gpuErrchk(hipDeviceSynchronize());

		#if TEST_FOR_EQUIVALENCE
			printf("q,qd,qdd,u\n");
			printMat<T,1,NUM_POS>(h_x,1);
			printMat<T,1,NUM_POS>(&h_x[NUM_POS],1);
			printMat<T,1,NUM_POS>(h_qdd,1);
			printMat<T,1,NUM_POS>(h_u,1);
			printf("Minv\n");
			printMat<T,NUM_POS,NUM_POS>(h_Minv,NUM_POS);
			dynamicsGradient_v2<T,VEL_DAMPING>(h_dqdd,h_qdd,h_x,h_u,h_Minv,h_I,h_T);
			printf("dqdd_dq\n");
			printMat<T,NUM_POS,NUM_POS>(h_dqdd,NUM_POS);
			printf("dqdd_dqd\n");
			printMat<T,NUM_POS,NUM_POS>(&h_dqdd[NUM_POS*NUM_POS],NUM_POS);
		#endif


		clock_gettime(CLOCK_MONOTONIC,&start);
		kern_single<T,SINGLE_TEST_ITERS,VEL_DAMPING><<<1,dimms>>>(d_dqdd,d_qdd,d_x,d_u,d_Minv,d_I,d_T);
		gpuErrchk(hipDeviceSynchronize());
		clock_gettime(CLOCK_MONOTONIC,&end);
		printf("Single Call vaf+dc/du+dqdd/du %fus\n",time_delta_us_timespec(start,end)/static_cast<double>(SINGLE_TEST_ITERS));

		#if TEST_FOR_EQUIVALENCE
			gpuErrchk(hipMemcpy(h_dqdd,d_dqdd,3*NUM_POS*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T),hipMemcpyDeviceToHost));
			printf("dqdd_dq - kernel\n");
			printMat<T,NUM_POS,NUM_POS>(h_dqdd,NUM_POS);
			printf("dqdd_dqd - kernel\n");
			printMat<T,NUM_POS,NUM_POS>(&h_dqdd[NUM_POS*NUM_POS],NUM_POS);
		#endif

		clock_gettime(CLOCK_MONOTONIC,&start);
		kern_single_vaf_dcdu<T,SINGLE_TEST_ITERS,VEL_DAMPING><<<1,dimms>>>(d_dqdd,d_qdd,d_x,d_u,d_Minv,d_I,d_T);
		gpuErrchk(hipDeviceSynchronize());
		clock_gettime(CLOCK_MONOTONIC,&end);
		printf("Single Call vaf+dc/du %fus\n",time_delta_us_timespec(start,end)/static_cast<double>(SINGLE_TEST_ITERS));
		
		clock_gettime(CLOCK_MONOTONIC,&start);
		kern_single_vaf<T,SINGLE_TEST_ITERS,VEL_DAMPING><<<1,dimms>>>(d_dqdd,d_qdd,d_x,d_u,d_Minv,d_I,d_T);
		gpuErrchk(hipDeviceSynchronize());
		clock_gettime(CLOCK_MONOTONIC,&end);
		printf("Single Call vaf %fus\n",time_delta_us_timespec(start,end)/static_cast<double>(SINGLE_TEST_ITERS));
		printf("----------------------------------------\n");
	}
	else{
		std::vector<double> times = {};
	    for(int iter = 0; iter < TEST_ITERS; iter++){
	    	clock_gettime(CLOCK_MONOTONIC,&start);
		    gpuErrchk(hipMemcpy(d_mem_vol,h_mem_vol,(4*NUM_POS + NUM_POS*NUM_POS)*NUM_TIME_STEPS_TEST*sizeof(T),hipMemcpyHostToDevice));
			kern<T,NUM_TIME_STEPS_TEST,VEL_DAMPING><<<NUM_TIME_STEPS_TEST,dimms>>>(d_dqdd2,d_mem_vol,d_mem_const);
			gpuErrchk(hipDeviceSynchronize());
			gpuErrchk(hipMemcpy(h_dqdd2,d_dqdd2,3*NUM_POS*NUM_POS*NUM_TIME_STEPS_TEST*sizeof(T),hipMemcpyDeviceToHost));
			clock_gettime(CLOCK_MONOTONIC,&end);
			times.push_back(time_delta_us_timespec(start,end));
		}
		printf("[N=%d] GPU Compute + I/O: ",NUM_TIME_STEPS_TEST); printStats<PRINT_DISTRIBUTIONS_GLOBAL>(&times);
		std::vector<double> times2 = {};
	    for(int iter = 0; iter < TEST_ITERS; iter++){
	    	clock_gettime(CLOCK_MONOTONIC,&start);
		    kern<T,NUM_TIME_STEPS_TEST,VEL_DAMPING><<<NUM_TIME_STEPS_TEST,dimms>>>(d_dqdd2,d_mem_vol,d_mem_const);
			gpuErrchk(hipDeviceSynchronize());
			clock_gettime(CLOCK_MONOTONIC,&end);
			times2.push_back(time_delta_us_timespec(start,end));
		}
		printf("[N:%d] GPU Compute: ",NUM_TIME_STEPS_TEST); printStats<PRINT_DISTRIBUTIONS_GLOBAL>(&times2);
		printf("----------------------------------------\n");
	}

    // free all
	free(h_qdd);
	free(h_dqdd);
	free(h_dqdd2);
	free(h_Minv);
	free(h_x);
	free(h_u);
	free(h_I);
	free(h_T);
	gpuErrchk(hipFree(d_qdd));
	gpuErrchk(hipFree(d_dqdd));
	gpuErrchk(hipFree(d_Minv));
	gpuErrchk(hipFree(d_x));
	gpuErrchk(hipFree(d_u));
	gpuErrchk(hipFree(d_I));
	gpuErrchk(hipFree(d_T));
	for(int i=0; i<NUM_STREAMS; i++){gpuErrchk(hipStreamDestroy(streams[i]));} free(streams);
}

int main(void){
	test<float,TEST_ITERS_GLOBAL,1>();
	#if !TEST_FOR_EQUIVALENCE
		test<float,TEST_ITERS_GLOBAL,10>();
		test<float,TEST_ITERS_GLOBAL,16>();
		test<float,TEST_ITERS_GLOBAL,32>();
		test<float,TEST_ITERS_GLOBAL,64>();
		test<float,TEST_ITERS_GLOBAL,128>();
		test<float,TEST_ITERS_GLOBAL,256>();
	#endif
}
